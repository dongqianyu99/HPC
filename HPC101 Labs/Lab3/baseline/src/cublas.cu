#include <hipblaslt.h>
#include <hip/hip_runtime_api.h>

#include "common.cuh"
#include "gemm.cuh"
#include <cute/tensor.hpp>

template <>
void run_cublas<float>(thrust::device_vector<float>& d_A,
                       thrust::device_vector<float>& d_B,
                       thrust::device_vector<float>& d_C, const int m,
                       const int n, const int k) {

    hipblasLtHandle_t ltHandle = NULL;
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    const int lda = k;
    const int ldb = k;
    const int ldc = m;

    thrust::device_vector<float> d_workspace(1024 * 1024 * 4);
    float* workspace = d_workspace.data().get();
    size_t workspaceSize = d_workspace.size() * sizeof(float);
    float* A = d_A.data().get();
    float* B = d_B.data().get();
    float* C = d_C.data().get();

    CUBLAS_CALL(hipblasLtCreate(&ltHandle));


    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    CUBLAS_CALL(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    CUBLAS_CALL(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    CUBLAS_CALL(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    CUBLAS_CALL(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    CUBLAS_CALL(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    CUBLAS_CALL(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));

    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from hipMalloc)
    CUBLAS_CALL(hipblasLtMatmulPreferenceCreate(&preference));
    CUBLAS_CALL(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    CUBLAS_CALL(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        CUBLAS_CALL(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    CUBLAS_CALL(hipblasLtMatmul(ltHandle,
                               operationDesc,
                               &alpha,
                               A,
                               Adesc,
                               B,
                               Bdesc,
                               &beta,
                               C,
                               Cdesc,
                               C,
                               Cdesc,
                               &heuristicResult.algo,
                               workspace,
                               workspaceSize,
                               0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (ltHandle) CUBLAS_CALL(hipblasLtDestroy(ltHandle));
    if (preference) CUBLAS_CALL(hipblasLtMatmulPreferenceDestroy(preference));
    if (Cdesc) CUBLAS_CALL(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) CUBLAS_CALL(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) CUBLAS_CALL(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) CUBLAS_CALL(hipblasLtMatmulDescDestroy(operationDesc));
}

template <>
void run_cublas<cute::half_t>(thrust::device_vector<cute::half_t>& d_A,
                       thrust::device_vector<cute::half_t>& d_B,
                       thrust::device_vector<cute::half_t>& d_C, const int m,
                       const int n, const int k) {

    hipblasLtHandle_t ltHandle = NULL;
    const cute::half_t alpha(1.0f);
    const cute::half_t beta(0.0f);
    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    const int lda = k;
    const int ldb = k;
    const int ldc = m;

    thrust::device_vector<int> d_workspace(1024 * 1024);
    void* workspace = d_workspace.data().get();
    size_t workspaceSize = d_workspace.size() * sizeof(int);
    cute::half_t* A = d_A.data().get();
    cute::half_t* B = d_B.data().get();
    cute::half_t* C = d_C.data().get();

    CUBLAS_CALL(hipblasLtCreate(&ltHandle));


    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    CUBLAS_CALL(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_16F, HIP_R_16F));
    CUBLAS_CALL(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    CUBLAS_CALL(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    CUBLAS_CALL(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    CUBLAS_CALL(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    CUBLAS_CALL(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16F, m, n, ldc));

    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from hipMalloc)
    CUBLAS_CALL(hipblasLtMatmulPreferenceCreate(&preference));
    CUBLAS_CALL(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    CUBLAS_CALL(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) {
        CUBLAS_CALL(HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    CUBLAS_CALL(hipblasLtMatmul(ltHandle,
                               operationDesc,
                               &alpha,
                               A,
                               Adesc,
                               B,
                               Bdesc,
                               &beta,
                               C,
                               Cdesc,
                               C,
                               Cdesc,
                               &heuristicResult.algo,
                               workspace,
                               workspaceSize,
                               0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (ltHandle) CUBLAS_CALL(hipblasLtDestroy(ltHandle));
    if (preference) CUBLAS_CALL(hipblasLtMatmulPreferenceDestroy(preference));
    if (Cdesc) CUBLAS_CALL(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) CUBLAS_CALL(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) CUBLAS_CALL(hipblasLtMatrixLayoutDestroy(Adesc));
    if (operationDesc) CUBLAS_CALL(hipblasLtMatmulDescDestroy(operationDesc));
}